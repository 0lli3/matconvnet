// @file datacu.cu
// @brief Basic data structures (CUDA support)
// @author Andrea Vedaldi

/*
Copyright (C) 2015 Andrea Vedaldi.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#ifndef ENABLE_GPU
#error "datacu.cu cannot be compiled without GPU support"
#endif

#include "datacu.hpp"
#include "impl/blashelper.hpp"
#include <hipblas.h>

using namespace vl ;

/* -------------------------------------------------------------------
 * CudaHelper
 * ---------------------------------------------------------------- */

vl::CudaHelper::CudaHelper()
: isCublasInitialized(false)
#if ENABLE_CUDNN
, isCudnnInitialized(false), cudnnEnabled(true)
#endif
{ }

vl::CudaHelper::~CudaHelper()
{
  clearCublas() ;
#ifdef ENABLE_CUDNN
  clearCudnn() ;
#endif
}

/* -------------------------------------------------------------------
 * getCublasHandle
 * ---------------------------------------------------------------- */

#include <iostream>

hipblasStatus_t vl::CudaHelper::getCublasHandle(hipblasHandle_t* handle)
{
  bool ok = isCublasInitialized ;
  if (ok) {
    int gpu ;
    hipGetDevice(&gpu) ;
    ok &= (gpu == cublasDeviceId) ;
    std::cout<<"CudaHelper:: new CUBLAS device detected "<<gpu<<std::endl ;
  }
  if (!ok) {
    clearCublas() ;
    hipblasStatus_t stat = hipblasCreate(&cublasHandle) ;
    if (stat != HIPBLAS_STATUS_SUCCESS) { return stat ; }
    isCublasInitialized = true ;
    hipGetDevice(&cublasDeviceId) ;
  }
  *handle = cublasHandle ;
  return HIPBLAS_STATUS_SUCCESS ;
}

void vl::CudaHelper::clearCublas()
{
  if (!isCublasInitialized) { return ; }
  int gpu ;
  hipGetDevice(&gpu) ;
  bool needSwitch = (gpu != cublasDeviceId) ;
  if (needSwitch) {
    std::cout<<"CudaHelper:: switch to "<<cublasDeviceId<<std::endl ;
    hipError_t status = hipSetDevice(cublasDeviceId) ;
    std::cout<<"CudaHelper:: switched to "<<cublasDeviceId<<" "<<status<<std::endl ;
  }
  std::cout<<"CudaHelper:: destroying "<<std::endl ;
  hipblasDestroy(cublasHandle) ;  std::cout<<"CudaHelper:: destroyed "<<std::endl ;
  isCublasInitialized = false ;
  cublasDeviceId = -1 ;
  if (needSwitch) {
    std::cout<<"CudaHelper: switch back to "<<gpu<<std::endl ;
    hipSetDevice(gpu) ;
  }
}

/* -------------------------------------------------------------------
 * getCudnnHandle
 * ---------------------------------------------------------------- */

#if ENABLE_CUDNN
hipdnnStatus_t
vl::CudaHelper::getCudnnHandle(hipdnnHandle_t* handle)
{
  bool ok = isCudnnInitialized ;
  if (ok) {
    int gpu ;
    hipGetDevice(&gpu) ;
    ok &= (gpu == cudnnDeviceId) ;
  }
  if (!ok) {
    clearCudnn() ;
    hipdnnStatus_t stat = hipdnnCreate(&cudnnHandle) ;
    if (stat != HIPDNN_STATUS_SUCCESS) { return stat ; }
    isCudnnInitialized = true ;
    hipGetDevice(&cudnnDeviceId) ;
  }
  *handle = cudnnHandle ;
  return HIPDNN_STATUS_SUCCESS ;
}

void
vl::CudaHelper::clearCudnn()
{
  if (!isCudnnInitialized) { return ; }
  int gpu ;
  hipGetDevice(&gpu) ;
  bool needSwitch = (gpu != cudnnDeviceId) ;
  if (needSwitch) { hipSetDevice(cudnnDeviceId) ; }
  hipdnnDestroy(cudnnHandle) ;
  isCudnnInitialized = false ;
  cudnnDeviceId = -1 ;
  if (needSwitch) { hipSetDevice(gpu) ; }
}

bool
vl::CudaHelper::getCudnnEnabled() const
{
  return cudnnEnabled ;
}

void
vl::CudaHelper::setCudnnEnabled(bool active)
{
  cudnnEnabled = active ;
}
#endif

/* -------------------------------------------------------------------
 * CuBLAS Errors
 * ---------------------------------------------------------------- */

static const char *
getCublasErrorMessageFromStatus(hipblasStatus_t status)
{
  switch (status) {
    case HIPBLAS_STATUS_SUCCESS: return "HIPBLAS_STATUS_SUCCESS";
    case HIPBLAS_STATUS_NOT_INITIALIZED: return "HIPBLAS_STATUS_NOT_INITIALIZED";
    case HIPBLAS_STATUS_ALLOC_FAILED: return "HIPBLAS_STATUS_ALLOC_FAILED";
    case HIPBLAS_STATUS_INVALID_VALUE: return "HIPBLAS_STATUS_INVALID_VALUE";
    case HIPBLAS_STATUS_ARCH_MISMATCH: return "HIPBLAS_STATUS_ARCH_MISMATCH";
    case HIPBLAS_STATUS_MAPPING_ERROR: return "HIPBLAS_STATUS_MAPPING_ERROR";
    case HIPBLAS_STATUS_EXECUTION_FAILED: return "HIPBLAS_STATUS_EXECUTION_FAILED";
    case HIPBLAS_STATUS_INTERNAL_ERROR: return "HIPBLAS_STATUS_INTERNAL_ERROR";
#if CUDA_VERSION >= 6000
    case HIPBLAS_STATUS_NOT_SUPPORTED: return "HIPBLAS_STATUS_NOT_SUPPORTED";
#endif
#if CUDA_VERSION >= 6050
    case HIPBLAS_STATUS_UNKNOWN: return "HIPBLAS_STATUS_UNKNOWN";
#endif
  }
  return "CuBLAS unknown status" ;
}

vl::Error
vl::CudaHelper::catchCublasError(hipblasStatus_t status, char const * description)
{
  /* if there is no CuBLAS error, do not do anything */
  if (status == HIPBLAS_STATUS_SUCCESS) { return vl::vlSuccess ; }

  /* if there is a CuBLAS error, store it */
  lastCublasError = status ;
  std::string message = getCublasErrorMessageFromStatus(status) ;
  if (description) {
    message = std::string(description) + " (" + message + ")" ;
  }
  lastCublasErrorMessage = message ;
  return vl::vlErrorCublas ;
}

hipblasStatus_t
vl::CudaHelper::getLastCublasError() const
{
  return lastCublasError;
}

std::string const&
vl::CudaHelper::getLastCublasErrorMessage() const
{
  return lastCublasErrorMessage ;
}

/* -------------------------------------------------------------------
 * CuDNN Errors
 * ---------------------------------------------------------------- */

#if ENABLE_CUDNN
vl::Error
vl::CudaHelper::catchCudnnError(hipdnnStatus_t status, char const* description)
{
  /* if there is no CuDNN error, do not do anything */
  if (status == HIPDNN_STATUS_SUCCESS) { return vl::vlSuccess ; }

  /* if there is a CuDNN error, store it */
  lastCudnnError = status ;
  std::string message = hipdnnGetErrorString(status) ;
  if (description) {
    message = std::string(description) + " (" + message + ")" ;
  }
  lastCudnnErrorMessage = message ;
  return vl::vlErrorCudnn ;
}

hipdnnStatus_t
vl::CudaHelper::getLastCudnnError() const
{
  return lastCudnnError;
}

std::string const&
vl::CudaHelper::getLastCudnnErrorMessage() const
{
  return lastCudnnErrorMessage ;
}
#endif

/* -------------------------------------------------------------------
 * Cuda Errors
 * ---------------------------------------------------------------- */

vl::Error
vl::CudaHelper::catchCudaError(char const* description)
{
  /* if there is no Cuda error, do not do anything */
  hipError_t error = hipPeekAtLastError() ;
  if (error == hipSuccess) { return vl::vlSuccess ; }

  /* if there is a Cuda error, eat it and store it */
  lastCudaError = hipGetLastError() ;
  std::string message = hipGetErrorString(lastCudaError) ;
  if (description) {
    message = std::string(description) + ": " + message ;
  }
  lastCudaErrorMessage = message ;
  return vl::vlErrorCuda ;
}

hipError_t
vl::CudaHelper::getLastCudaError() const
{
  return lastCudaError ;
}

std::string const&
vl::CudaHelper::getLastCudaErrorMessage() const
{
  return lastCudaErrorMessage ;
}



