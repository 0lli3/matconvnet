//
//  datacu.cpp
//  matconv
//
//  Created by Andrea Vedaldi on 09/02/2015.
//  Copyright (c) 2015 Andrea Vedaldi. All rights reserved.
//

#include "datacu.hpp"

using namespace vl ;

#ifndef ENABLE_GPU
#error "datacu.cpp cannot be compiled without GPU support"
#endif

/* -------------------------------------------------------------------
 * CudaHelper
 * ---------------------------------------------------------------- */

vl::CudaHelper::CudaHelper()
: isCuBLASInitialized(false)
#if ENABLE_CUDNN
, isCuDNNInitialized(false), cudnnActive(true)
#endif
{ }

vl::CudaHelper::~CudaHelper()
{
  if (isCuBLASInitialized) {
    hipblasDestroy(cuBLASHandle) ;
    isCuBLASInitialized = false ;
  }
#ifdef ENABLE_CUDNN
  if (isCuDNNInitialized) {
    hipdnnDestroy(cuDNNHandle) ;
    isCuDNNInitialized = false ;
  }
#endif
}

/* -------------------------------------------------------------------
 * getCuBLASHandle
 * ---------------------------------------------------------------- */

hipblasStatus_t vl::CudaHelper::getCuBLASHandle(hipblasHandle_t* handle)
{
  if (!isCuBLASInitialized) {
    hipblasStatus_t stat = hipblasCreate(&cuBLASHandle) ;
    if (stat != HIPBLAS_STATUS_SUCCESS) { return stat ; }
    isCuBLASInitialized = true ;
  }
  *handle = cuBLASHandle ;
  return HIPBLAS_STATUS_SUCCESS ;
}

/* -------------------------------------------------------------------
 * getCuDNNHandle
 * ---------------------------------------------------------------- */

#if ENABLE_CUDNN
hipdnnStatus_t vl::CudaHelper::getCuDNNHandle(hipdnnHandle_t* handle)
{
  if (!isCuDNNInitialized) {
    hipdnnStatus_t stat = hipdnnCreate(&cuDNNHandle) ;
    if (stat != HIPDNN_STATUS_SUCCESS) { return stat ; }
    isCuDNNInitialized = true ;
  }
  *handle = cuDNNHandle ;
  return HIPDNN_STATUS_SUCCESS ;
}

bool vl::CudaHelper::isCudnnActive() const
{
  return cudnnActive ;
}

void vl::CudaHelper::setCudnnActive(bool active)
{
  cudnnActive = active ;
}
#endif



