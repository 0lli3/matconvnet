/** @file gconv.cu
 ** @brief Convolution block
 ** @author Andrea Vedaldi
 **/

#include "mex.h"
#ifdef ENABLE_GPU
#include "gpu/mxGPUArray.h"
#endif
#include "bits/mexutils.h"
#include "bits/im2col.hpp"

#include <blas.h>
#include <iostream>
#include <assert.h>

#ifdef ENABLE_GPU
#include <hipblas.h>
#endif

/* option codes */
enum {
  opt_stride = 0,
  opt_pad,
  opt_verbose
} ;

/* options */
vlmxOption  options [] = {
  {"Stride",           1,   opt_stride            },
  {"Pad",              1,   opt_pad               },
  {"Verbose",          0,   opt_verbose           },
  {0,                  0,   0                     }
} ;

/* ---------------------------------------------------------------- */
/*                                                 Helper functions */
/* ---------------------------------------------------------------- */

typedef struct FeatureMapGeometry_
{
  mxClassID classID ;
  ptrdiff_t height ;
  ptrdiff_t width ;
  ptrdiff_t depth ;
  ptrdiff_t size ;
  ptrdiff_t numElements ;
} FeatureMapGeometry ;

typedef struct FeatureMap_
{
  bool isOwner ;
  mxArray * array ;
#ifdef ENABLE_GPU
  mxGPUArray * gpuArray ;
#endif
  FeatureMapGeometry geom ;
} FeatureMap ;

void feature_map_display (FeatureMap const * map, char const * name)
{
  double const MB = 1024.0 * 1024.0 ;
  mexPrintf("vl_nnconv: %s: %d x %d x %d x %d [%.1f MB]\n",
            name,
            map->geom.height, map->geom.width, map->geom.depth, map->geom.size,
            (double)(map->geom.numElements*4)/MB) ;
}

void feature_map_init_with_array (FeatureMap * map, bool gpuMode, mxArray const* array)
{
  mwSize const * dimensions ;
  mwSize numDimensions ;

#ifndef ENABLE_GPU
  assert(!gpuMode) ;
#endif

  map->isOwner = false ;
  map->array = (mxArray*)array ;
#ifdef ENABLE_GPU
  map->gpuArray = NULL ;
  if (gpuMode) {
    if (!mxIsGPUArray(map->array)) {
      mexErrMsgTxt("The inputs are of mixed GPU and CPU types.") ;
    }
    map->gpuArray = (mxGPUArray*) mxGPUCreateFromMxArray(map->array) ;
    map->geom.classID = mxGPUGetClassID(map->gpuArray) ;
    map->geom.numElements = mxGPUGetNumberOfElements(map->gpuArray) ;
    dimensions = mxGPUGetDimensions(map->gpuArray) ;
    numDimensions = mxGPUGetNumberOfDimensions(map->gpuArray) ;
  } else
#endif
  {
    if (!mxIsNumeric(map->array)) {
      mexErrMsgTxt("The inputs are neither all numeric CPU arrays or GPU arrays.") ;
    }
    map->geom.classID = mxGetClassID(map->array) ;
    map->geom.numElements = mxGetNumberOfElements(map->array) ;
    dimensions = mxGetDimensions(map->array) ;
    numDimensions = mxGetNumberOfDimensions(map->array) ;
  }
  map->geom.height = (numDimensions >= 1) ? dimensions[0] : 1 ;
  map->geom.width = (numDimensions >= 2) ? dimensions[1] : 1 ;
  map->geom.depth = (numDimensions >= 3) ? dimensions[2] : 1 ;
  map->geom.size = (numDimensions >= 4) ? dimensions[3] : 1 ;
}

void feature_map_init_with_geom (FeatureMap * map, bool gpuMode,
                                 FeatureMapGeometry geom, bool initialize)
{
  mwSize dimensions [4] = {geom.height, geom.width, geom.depth, geom.size} ;
  map->isOwner = true ;
  map->geom = geom ;
  map->array = NULL ;
#ifdef ENABLE_GPU
  map->gpuArray = NULL ;
  if (gpuMode) {
    map->gpuArray = mxGPUCreateGPUArray(4, dimensions, mxSINGLE_CLASS, mxREAL,
                                        (initialize)?MX_GPU_INITIALIZE_VALUES:MX_GPU_DO_NOT_INITIALIZE) ;
  } else
#endif
  {
    map->array = mxCreateNumericArray(4, dimensions, mxSINGLE_CLASS, mxREAL) ;
  }
}

void feature_map_init_with_geom_and_ones (FeatureMap * map, bool gpuMode,
                                          FeatureMapGeometry geom)
{
  mwSize dimensions [4] = {geom.height, geom.width, geom.depth, geom.size} ;
  map->isOwner = true ;
  map->geom = geom ;
  map->array = mxCreateNumericArray(4, dimensions, mxSINGLE_CLASS, mxREAL) ;
  int i ;
  float* data = (float*)mxGetData(map->array) ;
  for (i = 0 ; i < geom.numElements ; ++i) { data[i] = 1.0f ; }
#ifdef ENABLE_GPU
  map->gpuArray = NULL ;
  if (gpuMode) {
    map->gpuArray = (mxGPUArray*) mxGPUCreateFromMxArray (map->array) ;
  }
#endif
}

void feature_map_deinit (FeatureMap * map)
{
#ifdef ENABLE_GPU
  if (map->gpuArray) {
    mxGPUDestroyGPUArray(map->gpuArray) ;
    map->gpuArray = NULL ;
  }
#endif
  if (map->isOwner && map->array) {
    mxDestroyArray(map->array) ;
  }
  map->array = NULL ;
}

mxArray* feature_map_deinit_extracting_array(FeatureMap * map)
{
  mxArray* array = map->array ;
  map->array = NULL ;
#ifdef ENABLE_GPU
  if (map->gpuArray) {
    if (!array) {
      array = mxGPUCreateMxArrayOnGPU(map->gpuArray) ;
    }
    mxGPUDestroyGPUArray(map->gpuArray) ;
    map->gpuArray = NULL ;
  }
#endif
  return array ;
}

/* ---------------------------------------------------------------- */
/*                                                       MEX driver */
/* ---------------------------------------------------------------- */

enum {
  IN_DATA = 0, IN_FILTERS, IN_BIASES, IN_DEROUTPUT, IN_END
} ;

enum {
  OUT_RESULT = 0, OUT_DERFILTERS, OUT_DERBIASES, OUT_END
} ;

void mexFunction(int nout, mxArray *out[],
                 int nin, mxArray const *in[])
{
  /* inputs */
  FeatureMap data ;
  FeatureMap filters ;
  FeatureMap biases ;
  FeatureMap derOutput ;

  /* temporary buffers */
  FeatureMap temp ;
  FeatureMap allOnes ;

  /* outputs */
  FeatureMap output ;
  FeatureMap derData ;
  FeatureMap derFilters ;
  FeatureMap derBiases ;

  int stride = 1 ;
  int pad = 0 ;
  int numGroups = 1 ;

#if ENABLE_GPU
  hipblasStatus_t stat;
  hipblasHandle_t handle;
  bool gpuMode = false ;
#else
  bool const gpuMode = false ;
#endif
  bool backMode = false ;
  bool biasMode = false ;
  bool fullyConnectedMode = false ;
  bool computeOutput = true ;
  bool computeDerData = true ;
  bool computeDerFilters = true ;
  bool computeDerBiases = true ;

  int verbosity = 0 ;
  int opt ;
  int next = IN_END ;
  mxArray const *optarg ;

  /* -------------------------------------------------------------- */
  /*                                            Check the arguments */
  /* -------------------------------------------------------------- */

  if (nin < 3) {
    mexErrMsgTxt("There are less than three arguments.") ;
  }

  if (nin > 3 && vlmxIsString(in[3],-1)) {
    next = 3 ;
    backMode = 0 ;
  } else {
    backMode = (nin >= 4) ;
  }

  biasMode = (mxGetNumberOfElements(in[IN_BIASES]) > 0) ;

#if ENABLE_GPU
  gpuMode = mxIsGPUArray(in[IN_DATA]) ;
  if (gpuMode) {
    mxInitGPU() ;
    stat = hipblasCreate(&handle) ;
    if (stat != HIPBLAS_STATUS_SUCCESS) {
      mexErrMsgTxt("Could not initialize cuBLAS.") ;
    }
  }
#else
  if (!mxIsNumeric(in[IN_DATA])) {
    mexErrMsgTxt("DATA must be numeric (note: GPU support not compiled).") ;
  }
#endif

  while ((opt = vlmxNextOption (in, nin, options, &next, &optarg)) >= 0) {
    switch (opt) {
      case opt_verbose :
        ++ verbosity ;
        break ;

      case opt_stride :
        if (!vlmxIsPlainScalar(optarg) || (stride = (int) *mxGetPr(optarg)) < 1) {
          mexErrMsgTxt("STRIDE must be a positive integer.") ;
        }
        break ;

      case opt_pad :
        if (!vlmxIsPlainScalar(optarg) || (pad = (int) *mxGetPr(optarg)) < 0) {
          mexErrMsgTxt("PAD must be a non-negative integer.") ;
        }
        break ;

      default: break ;
    }
  }

  feature_map_init_with_array (&data, gpuMode, in[IN_DATA]) ;
  feature_map_init_with_array (&filters, gpuMode, in[IN_FILTERS]) ;
  if (biasMode) { feature_map_init_with_array(&biases, gpuMode, in[IN_BIASES]) ; }
  if (backMode) { feature_map_init_with_array(&derOutput, gpuMode, in[IN_DEROUTPUT]) ; }

  if (data.geom.classID != mxSINGLE_CLASS) {
    mexErrMsgTxt("DATA is not of class SINGLE.");
  }
  if (filters.geom.classID != mxSINGLE_CLASS) {
    mexErrMsgTxt("FILTERS is not of class SINGLE.");
  }
  if (biasMode && (biases.geom.classID != mxSINGLE_CLASS)) {
    mexErrMsgTxt("BIASES is not of class SINGLE.");
  }
  if (backMode && (derOutput.geom.classID != mxSINGLE_CLASS)) {
    mexErrMsgTxt("DEROUTPUT is not of class SINGLE.");
  }

  output.geom.height = (data.geom.height + 2*pad - filters.geom.height)/stride + 1 ;
  output.geom.width = (data.geom.width + 2*pad - filters.geom.width)/stride + 1 ;
  output.geom.depth = filters.geom.size ;
  output.geom.size = data.geom.size ;
  output.geom.numElements = output.geom.height*output.geom.width*output.geom.depth*output.geom.size ;
  derData.geom = data.geom ;
  derFilters.geom = filters.geom ;
  if (biasMode) {
    allOnes.geom.height = output.geom.height ;
    allOnes.geom.width = output.geom.width ;
    allOnes.geom.depth = 1 ;
    allOnes.geom.size = 1 ;
    allOnes.geom.numElements = allOnes.geom.height*allOnes.geom.width*allOnes.geom.depth*allOnes.geom.size ;
    derBiases.geom = biases.geom ;
  }

  /* grouped filters */
  numGroups = data.geom.depth / filters.geom.depth ;

  temp.geom.height = output.geom.height ;
  temp.geom.width = output.geom.width ;
  temp.geom.depth = filters.geom.height*filters.geom.width*filters.geom.depth*numGroups ;
  temp.geom.size = 1 ;

  /* if the output is 1x1 pixels, then there is no need to actually
   call im2col as it does not do anything
   */
  fullyConnectedMode = (output.geom.height == 1 && output.geom.width == 1) ;

  if (verbosity > 0) {
    mexPrintf("vl_nnconv: mode %s; %s\n", gpuMode?"gpu":"cpu", backMode?"backward":"forward") ;
    mexPrintf("vl_nnconv: stride: %d, pad: %d, numGroups: %d, bias: %d, fully connected: %d\n", stride, pad, numGroups, biasMode, fullyConnectedMode) ;
    feature_map_display(&data, "data") ;
    feature_map_display(&filters, "filters") ;
    if (biasMode) { feature_map_display(&biases, "biases") ; }
    if (backMode) {
      feature_map_display(&derOutput, "derOutput") ;
      feature_map_display(&derData, "derData") ;
      feature_map_display(&derFilters, "derFilters") ;
      if (biasMode) { feature_map_display(&derBiases, "derBiases") ; }
    } else {
      feature_map_display(&output, "output") ;
    }
    feature_map_display(&temp, "temp") ;
  }

  if (backMode) {
    if (derOutput.geom.height != temp.geom.height ||
        derOutput.geom.width != temp.geom.width ||
        derOutput.geom.depth != filters.geom.size ||
        derOutput.geom.size != data.geom.size)
    {
      mexErrMsgTxt("DEROUTPUT dimensions are incompatible with X and FILTERS.") ;
    }
  }

  if (numGroups * filters.geom.depth != data.geom.depth) {
    mexErrMsgTxt("The filter depth does not divide the image depth.") ;
  }

  if (filters.geom.size % numGroups != 0) {
    mexErrMsgTxt("The number of filter groups does not divide the total number of filters.") ;
  }

  if (data.geom.height + 2*pad < filters.geom.height || data.geom.width + 2*pad < filters.geom.width) {
    mexErrMsgTxt("FILTERS are larger than the DATA (including padding).") ;
  }

  if (filters.geom.height == 0 || filters.geom.width == 0 || filters.geom.depth == 0) {
    mexErrMsgTxt("A dimension of FILTERS is void.") ;
  }

  if (biasMode) {
    if (biases.geom.numElements != filters.geom.size) {
      mexErrMsgTxt("The number of elements of BIASES is not the same as the number of filters.") ;
    }
  }

  /* -------------------------------------------------------------- */
  /*                                                    Do the work */
  /* -------------------------------------------------------------- */

  if (biasMode) {
    feature_map_init_with_geom_and_ones(&allOnes, gpuMode, allOnes.geom) ;
  }
  if (!backMode && computeOutput) {
    feature_map_init_with_geom(&output, gpuMode, output.geom, false) ;
  }
  if (!fullyConnectedMode) {
    feature_map_init_with_geom (&temp, gpuMode, temp.geom, false);
  }
  if (backMode && computeDerData) {
    feature_map_init_with_geom(&derData, gpuMode, derData.geom, fullyConnectedMode) ;
  }
  if (backMode && computeDerFilters) {
    feature_map_init_with_geom(&derFilters, gpuMode, derFilters.geom, true) ;
  }
  if (backMode && biasMode && computeDerBiases) {
    feature_map_init_with_geom(&derBiases, gpuMode, derBiases.geom, true) ;
  }

  for (int image = 0 ; image < data.geom.size ; ++image) {
    /*
     temp (phi(x)): m x k
     filters, derFilters: k x n (for one group of filters)
     derOutput (dzdy) : m x n (for one group of filters)
     res (y) : m x n (for one group of filters)
     */
    ptrdiff_t dataOffset = (data.geom.height*data.geom.width*data.geom.depth) * image ;
    ptrdiff_t outputOffset = (output.geom.height*output.geom.width*output.geom.depth) * image ;
    ptrdiff_t derDataOffset = dataOffset ;
    ptrdiff_t derOutputOffset = outputOffset ;
    ptrdiff_t m = temp.geom.height * temp.geom.width ; /* num output pixels */
    ptrdiff_t n = filters.geom.size/numGroups ; /* num filters per group */
    ptrdiff_t k = filters.geom.height*filters.geom.width*filters.geom.depth ; /* filter volume */
    char OP_N = 'n' ;
    char OP_T = 't' ;

    if (backMode) {
      /* ---------------------------------------------------------- */
      /*                                              Backward mode */
      /* ---------------------------------------------------------- */

      /* compute derFilters dz/dF */
      if (computeDerFilters) {
        if (!fullyConnectedMode) {
          if (gpuMode) {
#ifdef ENABLE_GPU
            im2col_gpu<float>((float const*)mxGPUGetDataReadOnly(data.gpuArray) + dataOffset,
                              data.geom.depth, data.geom.width, data.geom.height,
                              filters.geom.width, filters.geom.height,
                              stride, pad,
                              (float *)mxGPUGetData(temp.gpuArray)) ;
#else
            assert(false) ;
#endif
          } else {
            im2col_cpu<float>((float const*)mxGetData(data.array) + dataOffset,
                              data.geom.depth, data.geom.width, data.geom.height,
                              filters.geom.width, filters.geom.height,
                              stride, pad,
                              (float *)mxGetData(temp.array)) ;
          }
        }
        for (int g = 0 ; g < numGroups ; ++ g) {
          ptrdiff_t filterGrpOffset = k * n * g ;
          ptrdiff_t tempGrpOffset = m * k * g ;
          ptrdiff_t derOutputGrpOffset = m * n * g ;
          float alpha = 1 ;
          float beta = 1 ;
          if (gpuMode) {
#ifdef ENABLE_GPU
            hipblasSgemm(handle,
                        HIPBLAS_OP_T, HIPBLAS_OP_N,
                        (int)k, (int)n, (int)m,
                        &alpha,
                        (float const*)mxGPUGetDataReadOnly(fullyConnectedMode ? data.gpuArray : temp.gpuArray)
                        + (fullyConnectedMode?dataOffset:0) + tempGrpOffset,
                        (int)m,
                        (float const*)mxGPUGetDataReadOnly(derOutput.gpuArray) + derOutputOffset + derOutputGrpOffset,
                        (int)m,
                        &beta,
                        (float*)mxGPUGetData(derFilters.gpuArray) + filterGrpOffset, (int)k) ;
#else
            assert(false) ;
#endif
          } else {
            sgemm(&OP_T, &OP_N,
                  &k, &n, &m,
                  &alpha,
                  (float*)mxGetData(fullyConnectedMode ? data.array : temp.array)
                  + (fullyConnectedMode?dataOffset:0) + tempGrpOffset, &m,
                  (float*)mxGetData(derOutput.array) + derOutputOffset + derOutputGrpOffset, &m,
                  &beta,
                  (float*)mxGetData(derFilters.array) + filterGrpOffset, &k) ;
          }
        }
      }

      /* compute derData dz/dbias */
      if (biasMode) {
        float alpha = 1 ;
        float beta = 1 ;
        ptrdiff_t q = filters.geom.size ;
        ptrdiff_t incx = 1 ;
        ptrdiff_t incy = 1 ;
        if (gpuMode) {
#ifdef ENABLE_GPU
          hipblasSgemv(handle,
                      HIPBLAS_OP_T,
                      (int)m, (int)q,
                      &alpha,
                      (float const*)mxGPUGetDataReadOnly(derOutput.gpuArray) + derOutputOffset, (int)m,
                      (float const*)mxGPUGetDataReadOnly(allOnes.gpuArray), (int)incx,
                      &beta,
                      (float*)mxGPUGetData(derBiases.gpuArray), (int)incy) ;
#else
          assert(false) ;
#endif
        } else {
          sgemv(&OP_T,
                &m, &q,
                &alpha,
                (float*)mxGetData(derOutput.array) + derOutputOffset, &m,
                (float*)mxGetData(allOnes.array), &incx,
                &beta,
                (float*)mxGetData(derBiases.array), &incy) ;
        }
      }

      /* compute derData dz/dx */
      if (computeDerData) {
        for (int g = 0 ; g < numGroups ; ++ g) {
          ptrdiff_t filterGrpOffset = k * n * g ;
          ptrdiff_t tempGrpOffset = m * k * g ;
          ptrdiff_t derOutputGrpOffset = m * n * g ;
          float alpha = 1 ;
          float beta = fullyConnectedMode ? 1 : 0 ;

          if (gpuMode) {
#ifdef ENABLE_GPU
            hipblasSgemm(handle,
                        HIPBLAS_OP_N, HIPBLAS_OP_T,
                        (int)m, (int)k, (int)n,
                        &alpha,
                        (float const*)mxGPUGetDataReadOnly(derOutput.gpuArray) + derOutputOffset + derOutputGrpOffset, (int)m,
                        (float const*)mxGPUGetDataReadOnly(filters.gpuArray) + filterGrpOffset, (int)k,
                        &beta,
                        (float*)mxGPUGetData(fullyConnectedMode ? derData.gpuArray : temp.gpuArray)
                        + (fullyConnectedMode ? + derDataOffset : 0) + tempGrpOffset,
                        (int)m) ;
#else
            assert(false) ;
#endif
          } else {
            sgemm(&OP_N, &OP_T,
                  &m, &k, &n,
                  &alpha,
                  (float*)mxGetData(derOutput.array) + derOutputOffset + derOutputGrpOffset, &m,
                  (float*)mxGetData(filters.array) + filterGrpOffset, &k,
                  &beta,
                  (float*)mxGetData(fullyConnectedMode ? derData.array : temp.array)
                  + (fullyConnectedMode ? + derDataOffset : 0) + tempGrpOffset,
                  &m) ;
          }
        }
        if (!fullyConnectedMode) {
          if (gpuMode) {
#ifdef ENABLE_GPU
            col2im_gpu<float>((float*)mxGPUGetDataReadOnly(temp.gpuArray),
                              data.geom.depth, data.geom.width, data.geom.height,
                              filters.geom.width, filters.geom.height,
                              stride, pad,
                              (float*)mxGPUGetData(derData.gpuArray) + derDataOffset) ;
#else
            assert(false) ;
#endif
          } else {
            col2im_cpu<float>((float*)mxGetData(temp.array),
                              data.geom.depth, data.geom.width, data.geom.height,
                              filters.geom.width, filters.geom.height,
                              stride, pad,
                              (float*)mxGetData(derData.array) + derDataOffset) ;
          }
        }
      }
    } else {
      /* ---------------------------------------------------------- */
      /*                                               Forward mode */
      /* ---------------------------------------------------------- */
      if (computeOutput) {
        if (!fullyConnectedMode) {
          if (gpuMode) {
#ifdef ENABLE_GPU
            im2col_gpu<float>((float const*)mxGPUGetDataReadOnly(data.gpuArray) + dataOffset,
                              data.geom.depth, data.geom.width, data.geom.height,
                              filters.geom.width, filters.geom.height,
                              stride, pad,
                              (float *)mxGPUGetData(temp.gpuArray)) ;
#else
            assert(false) ;
#endif
          } else {
            im2col_cpu<float>((float const*)mxGetData(data.array) + dataOffset,
                              data.geom.depth, data.geom.width, data.geom.height,
                              filters.geom.width, filters.geom.height,
                              stride, pad,
                              (float *)mxGetData(temp.array)) ;
          }
        }
        for (int g = 0 ; g < numGroups ; ++ g) {
          ptrdiff_t filterGrpOffset = k * n * g ;
          ptrdiff_t tempGrpOffset = m * k * g ;
          ptrdiff_t outputGrpOffset = m * n * g  ;
          float alpha = 1 ;
          float beta = 0 ;
          if (gpuMode) {
#ifdef ENABLE_GPU
            hipblasSgemm(handle,
                        HIPBLAS_OP_N, HIPBLAS_OP_N,
                        (int)m, (int)n, (int)k,
                        &alpha,
                        (float const*)mxGPUGetDataReadOnly(fullyConnectedMode ? data.gpuArray  : temp.gpuArray)
                        + (fullyConnectedMode?dataOffset:0) + tempGrpOffset, (int)m,
                        (float const*)mxGPUGetDataReadOnly(filters.gpuArray) + filterGrpOffset, (int)k,
                        &beta,
                        (float*)mxGPUGetData(output.gpuArray) + outputOffset + outputGrpOffset, (int)m) ;
#else
            assert(false) ;
#endif
          } else {
            sgemm(&OP_N, &OP_N,
                  &m, &n, &k,
                  &alpha,
                  (float*)mxGetData(fullyConnectedMode ? data.array : temp.array)
                  + (fullyConnectedMode?dataOffset:0) + tempGrpOffset, &m,
                  (float*)mxGetData(filters.array) + filterGrpOffset, &k,
                  &beta,
                  (float*)mxGetData(output.array) + outputOffset + outputGrpOffset, &m) ;
          }
        }
        if (biasMode) {
          float alpha = 1 ;
          float beta = 1 ;
          ptrdiff_t q = 1 ;
          if (gpuMode) {
#ifdef ENABLE_GPU
            hipblasSgemm(handle,
                        HIPBLAS_OP_N, HIPBLAS_OP_N,
                        (int)m, (int)biases.geom.numElements, (int)q,
                        &alpha,
                        (float const*)mxGPUGetDataReadOnly(allOnes.gpuArray) , (int)m,
                        (float const*)mxGPUGetDataReadOnly(biases.gpuArray), (int)q,
                        &beta,
                        (float*)mxGPUGetData(output.gpuArray) + outputOffset, (int)m) ;
#else
            assert(false) ;
#endif
          } else {
            sgemm(&OP_N, &OP_N,
                  &m, &biases.geom.numElements, &q,
                  &alpha,
                  (float*)mxGetData(allOnes.array), &m,
                  (float*)mxGetData(biases.array), &q,
                  &beta,
                  (float*)mxGetData(output.array) + outputOffset, &m) ;
          }
        }
      }
    }
  }

  /* -------------------------------------------------------------- */
  /*                                                        Cleanup */
  /* -------------------------------------------------------------- */
#ifdef ENABLE_GPU
  if (gpuMode) {
    hipblasDestroy(handle) ;
  }
#endif

  if (!fullyConnectedMode) { feature_map_deinit(&temp) ; }
  feature_map_deinit(&data) ;
  feature_map_deinit(&filters) ;
  if (biasMode) {
    feature_map_deinit(&biases) ;
    feature_map_deinit(&allOnes) ;
  }
  if (backMode) {
    out[OUT_RESULT] = feature_map_deinit_extracting_array(&derData) ;
    out[OUT_DERFILTERS] = feature_map_deinit_extracting_array(&derFilters) ;
    out[OUT_DERBIASES] = biasMode ? feature_map_deinit_extracting_array(&derBiases) : NULL ;
  } else {
    out[OUT_RESULT] = feature_map_deinit_extracting_array(&output) ;
  }
}
